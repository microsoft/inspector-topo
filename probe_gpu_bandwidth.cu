// Copyright (c) Microsoft Corporation.
// All rights reserved.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "probe_gpu_bandwidth.hpp"
#include <iostream>

int gpu_count() {
  int deviceCount;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  
  if (error_id != hipSuccess) {
    std::cerr << "hipGetDeviceCount returned " << error_id
              << ": " << hipGetErrorString(error_id)
              << std::endl;;
    exit(1);
  }

  return deviceCount;
}


double probe_gpu_bandwidth_from_numa_node(int numa_node, int gpuA, int gpuB) {
  std::cout << "Probing bandwidth to " << gpuA << " and " << gpuB
            << " from " << numa_node
            << std::endl;
  std::cout << "(not implemented)" << std::endl;

  // plan:
  // - allocate buffers on each GPU
  // - allocate buffers for each GPU on numa node
  // - enqueue N kernels copying from GPU to buffer
  // - time kernels
  // - compute aggregate bandwidth of copies
  
  return 0.0;
}
