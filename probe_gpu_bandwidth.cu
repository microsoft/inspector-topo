// Copyright (c) Microsoft Corporation.
// All rights reserved.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "probe_gpu_bandwidth.hpp"
#include <iostream>
#include <chrono>

#include <x86intrin.h>
#include <numa.h>
#include <gflags/gflags.h>

DECLARE_int64(length);
DEFINE_int32(bw_iters, 10, "Number of iterations to run when measuring GPU bandwidth.");

int gpu_count() {
  int deviceCount;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  
  if (error_id != hipSuccess) {
    std::cerr << "hipGetDeviceCount returned " << error_id
              << ": " << hipGetErrorString(error_id)
              << std::endl;;
    exit(1);
  }

  return deviceCount;
}


double probe_gpu_bandwidth_from_numa_node(int numa_nodeA, int gpuA, int numa_nodeB, int gpuB) {
  // std::cout << "Probing bandwidth with " <<
  // 	    << " for GPU doing " << gpuA << " on NUMA node " << numa_nodeA
  // 	    << " and GPU " << gpuB << " on NUMA node " << numa_nodeB
  //           << " from " << numa_node
  //           << std::endl;

  // plan:
  // - allocate buffers on each GPU
  // - allocate buffers for each GPU on numa node
  // - enqueue N kernels copying from GPU to buffer
  // - time kernels
  // - compute aggregate bandwidth of copies

  numa_run_on_node(numa_nodeA);  
  
  void * gpuA_host_buf = nullptr;
  void * gpuA_device_buf = nullptr;
  CHECK_CUDA(hipSetDevice(gpuA));
  CHECK_CUDA(hipHostAlloc(&gpuA_host_buf, FLAGS_length, hipHostMallocMapped));
  CHECK_CUDA(hipMalloc(&gpuA_device_buf, FLAGS_length));  

  hipStream_t gpuA_stream;
  CHECK_CUDA(hipStreamCreateWithFlags(&gpuA_stream, hipStreamNonBlocking));

  //numa_run_on_node(numa_nodeB);
  
  void * gpuB_host_buf = nullptr;
  void * gpuB_device_buf = nullptr;
  CHECK_CUDA(hipSetDevice(gpuB));
  CHECK_CUDA(hipHostAlloc(&gpuB_host_buf, FLAGS_length, hipHostMallocMapped));
  CHECK_CUDA(hipMalloc(&gpuB_device_buf, FLAGS_length));

  hipStream_t gpuB_stream;
  CHECK_CUDA(hipStreamCreateWithFlags(&gpuB_stream, hipStreamNonBlocking));

  // allow us to run anywhere
  //numa_run_on_node(-1);
    
  auto start_time = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < FLAGS_bw_iters; ++i) {
    CHECK_CUDA(hipSetDevice(gpuA));  
    CHECK_CUDA(hipMemcpyAsync(gpuA_host_buf, gpuA_device_buf, FLAGS_length, hipMemcpyDeviceToHost, gpuA_stream));
    CHECK_CUDA(hipSetDevice(gpuB));  
    //CHECK_CUDA(hipMemcpyAsync(gpuB_host_buf, gpuB_device_buf, FLAGS_length, hipMemcpyDeviceToHost, gpuB_stream));
    CHECK_CUDA(hipMemcpyAsync(gpuB_device_buf, gpuB_host_buf, FLAGS_length, hipMemcpyHostToDevice, gpuB_stream));    
  }

  CHECK_CUDA(hipSetDevice(gpuA));
  CHECK_CUDA(hipStreamSynchronize(gpuA_stream));
  CHECK_CUDA(hipSetDevice(gpuB));
  CHECK_CUDA(hipStreamSynchronize(gpuB_stream));
  auto end_time = std::chrono::high_resolution_clock::now();  

  // free memory
  CHECK_CUDA(hipHostFree(gpuA_host_buf));
  CHECK_CUDA(hipHostFree(gpuB_host_buf));
  CHECK_CUDA(hipFree(gpuA_device_buf));
  CHECK_CUDA(hipFree(gpuB_device_buf));
  
  uint64_t time_difference_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count();
  double bw = (double) FLAGS_length / (time_difference_ns / 1e9) / 1024 / 1024 / 1024 * FLAGS_bw_iters;
  std::cout << "Measured per-GPU bandwidth of " << bw
	    << " for GPU " << gpuA << " on NUMA node " << numa_nodeA << " doing DtoH" 
	    << " and GPU " << gpuB << " on NUMA node " << numa_nodeB << " doing HtoD" 
	    << std::endl;
  return bw;
}
