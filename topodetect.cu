#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// All rights reserved.

#include "probe_latency.hpp"
#include "LoopbackFlow.hpp"
#include "probe_gpu_bandwidth.hpp"

#include <gflags/gflags.h>
#include <numa.h>

#include <vector>
#include <limits>

DEFINE_int32(warmup,       100, "Number of warmup iterations to run before timing (defults to 100).");
DEFINE_int32(iters,       1000, "Number of timed iterations to run (defaults to 1000).");
DEFINE_int64(length, 1LL << 30, "Length of test buffers in bytes (defaults to 1 GB).");

int main(int argc, char * argv[]) {
  gflags::ParseCommandLineFlags(&argc, &argv, true);

  if (-1 == numa_available()) {
    std::cerr << "NUMA not available. Cannot probe topology." << std::endl;
    exit(1);
  }

  // Measure latency on each NUMA node. Remember the lowest.
  double min_latency = std::numeric_limits<double>::max();
  int min_numa_node = 0;
  for (int node = 0; node <= numa_max_node(); ++node) {
    double latency = probe_latency_from_numa_node(node);
    if (latency < min_latency) {
      min_latency = latency;
      min_numa_node = node;
    }
  }
  
  std::cout << "NIC appears to be nearest to NUMA node " << min_numa_node << std::endl;
  

  // Now probe bandwith to all pairs of GPUs while NIC is processing a
  // loopback flow.
  int gpu_count = 0;
  CHECK_CUDA(hipGetDeviceCount(&gpu_count));
  if (1 == gpu_count) {
    std::cout << "Simple case: single GPU. Not detecting NUMA node." << std::endl;
  } else {
    double gpu_pair_bw[gpu_count][gpu_count] = {0.0};

    // Track the pair with highest aggregate bandwidth.    
    double max_bw = std::numeric_limits<double>::min();
    int max_gpuA = 0;
    int max_gpuB = 1;

    // Track the pair with lowest aggregate bandwidth. This should be
    // the one shared with the NIC.
    double min_bw = std::numeric_limits<double>::max();
    int min_gpuA = 0;
    int min_gpuB = 1;

    { // Start loopback flow between NIC and the DRAM of the CPU it is nearest.
      LoopbackFlow flow(min_numa_node);
      
      for (int gpuA = 0; gpuA < gpu_count; ++gpuA) {
        for (int gpuB = gpuA + 1; gpuB < gpu_count; ++gpuB) {
          double bw = probe_gpu_bandwidth_from_numa_node(min_numa_node, gpuA, gpuB);
          gpu_pair_bw[gpuA][gpuB] = bw;
          
          // update best pair
          if (bw > max_bw) {
            max_bw = bw;
            max_gpuA = gpuA;
            max_gpuB = gpuB;
          }

          // update worst pair
          if (bw < min_bw) {
            min_bw = bw;
            min_gpuA = gpuA;
            min_gpuB = gpuB;
          }
        }
      }
    }

    std::cout << "Best GPU pair was " << max_gpuA << " and " << max_gpuB
              << " with a bandwidth of " << max_bw
              << std::endl;
    std::cout << "GPU pair shared with NIC appears to be " << min_gpuA << " and " << min_gpuB
              << " with a bandwidth of " << min_bw
              << std::endl;
  }

    
  std::cout << "Done." << std::endl;
  return 0;
}
